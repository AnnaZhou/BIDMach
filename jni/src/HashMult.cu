#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <MatKernel.hpp>

#if __CUDA_ARCH__ >= 300
#define MAXXGRID 2147483647
#else
#define MAXXGRID 65535
#endif



// Feature hashing multiply and multiply-transpose.
// This one enumerates, hashes and multiplies all pairs of features.
//
// NOTE: This uses a fast lookup recurrence which is only valid up to 3000 base features per column (approx 4.5 million pairs)


// Hash functions
// Adler32
__forceinline__ __device__ unsigned int adler32(const void *buf, size_t buflength) {
     const unsigned char *buffer = (const unsigned char*)buf;

     unsigned int s1 = 1;
     unsigned int s2 = 0;

     for (size_t n = 0; n < buflength; n++) {
        s1 = (s1 + buffer[n]) % 65521;
        s2 = (s2 + s1) % 65521;
     }     
     return (s2 << 16) | s1;
}

// MurmurHash3

static const unsigned int c1 = 0xcc9e2d51;
static const unsigned int c2 = 0x1b873593;
static const unsigned int r1 = 15;
static const unsigned int r2 = 13;
static const unsigned int m = 5;
static const unsigned int n = 0xe6546b64;

__forceinline__ __device__ unsigned int h1(unsigned int k, unsigned int hash) {

  k *= c1;
  k = (k << r1) | (k >> (32-r1));
  k *= c2;
 
  hash ^= k;
  hash = ((hash << r2) | (hash >> (32-r2)) * m) + n;
  return hash;
}

__forceinline__ __device__ unsigned int mmhash2(unsigned int v1, unsigned int v2, unsigned int mod)
{
  unsigned int hash = 3413413;
 
  hash = h1(v1, hash);
  hash = h1(v2, hash);
  
  hash ^= (hash >> 16);
  hash *= 0x85ebca6b;
  hash ^= (hash >> 13);
  hash *= 0xc2b2ae35;
  hash ^= (hash >> 16);
 
  return (hash % mod);
}

__forceinline__ __device__ int hash2(int a, int b, int modulus) {
  return mmhash2(a, b, modulus);
}

__forceinline__ __device__ int solve1(int j) {
  float v = sqrtf((float)j);
#pragma unroll
  for (int k = 0; k < 5; k++) {
    v = v - (v*(v+1)-2*j)/(2*v+1);   // Newton iterations to find first index. 
  }
  return (int)(v+2e-5f);   
}

__global__ void __hashmult(int nrows, int nfeats, int ncols, float *A, float *Bdata, int *Bir, int *Bjc, float *C) {
  int istart = ((long long)blockIdx.x) * ncols/ gridDim.x;
  int iend = ((long long)(blockIdx.x + 1)) * ncols / gridDim.x;
  for (int i = istart; i < iend ; i++) {                     // i is the column index
    int jstart = Bjc[i];                                     // Range of nz rows in this column
    int jend = Bjc[i+1];
    int nr = jend - jstart;                                  // Number of nz rows
    int todo = nr * (nr + 1) / 2;                            // Number of pairs to process (including k,k pairs)
    for (int j = threadIdx.y; j < todo; j += blockDim.y) {   // j indexes a worker for this column
      int j1 = solve1(j);                                    // Compute the first and second indices
      int j2 = j - j1*(j1+1)/2; 
      float f1 = Bdata[jstart + j1];                         // Get the two features
      float f2 = Bdata[jstart + j2];
      int r1 = Bir[jstart + j1];                             // And their row indices
      int r2 = Bir[jstart + j2];
      int ind = hash2(r1, r2, nfeats);                       // Hash the indices

      float sum = A[threadIdx.x + nrows * ind] * f1 * f2;    // Do the product
      atomicAdd(&C[threadIdx.x + nrows * i], sum);
    }
  }
}

int hashmult(int nrows, int nfeats, int ncols, float *A, float *Bdata, int *Bir, int *Bjc, float *C) {
  int nt = max(1, 256/nrows);
  dim3 threadDim(nrows, nt, 1);
  int nblocks = min(MAXXGRID, ncols);
  __hashmult<<<nblocks,threadDim>>>(nrows, nfeats, ncols, A, Bdata, Bir, Bjc, C);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  return err;
}


__global__ void __hashmultT(int nrows, int nfeats, int ncols, float *A, float *Bdata, int *Bir, int *Bjc, float *C) {
  int istart = ((long long)blockIdx.x) * ncols/ gridDim.x;
  int iend = ((long long)(blockIdx.x + 1)) * ncols / gridDim.x;
  for (int i = istart; i < iend ; i++) {                     // i is the column index
    int jstart = Bjc[i];                                     // Range of nz rows in this column
    int jend = Bjc[i+1];
    int nr = jend - jstart;                                  // Number of nz rows
    int todo = nr * (nr + 1) / 2;                            // Number of pairs to process (including k,k pairs)
    for (int j = threadIdx.y; j < todo; j += blockDim.y) {   // j indexes a worker for this column
      int j1 = solve1(j);                                    // Compute the first and second indices
      int j2 = j - j1*(j1+1)/2; 
      float f1 = Bdata[jstart + j1];                         // Get the two features
      float f2 = Bdata[jstart + j2];
      int r1 = Bir[jstart + j1];                             // And their row indices
      int r2 = Bir[jstart + j2];
      int ind = hash2(r1, r2, nfeats);                       // Hash the indices

      float sum = A[threadIdx.x + nrows * i] * f1 * f2;      // Do the product
      atomicAdd(&C[threadIdx.x + nrows * ind], sum);
    }
  }
}

int hashmultT(int nrows, int nfeats, int ncols, float *A, float *Bdata, int *Bir, int *Bjc, float *C) {
  int nt = max(1, 256/nrows);
  dim3 threadDim(nrows, nt, 1);
  int nblocks = min(MAXXGRID, ncols);
  __hashmultT<<<nblocks,threadDim>>>(nrows, nfeats, ncols, A, Bdata, Bir, Bjc, C);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  return err;
}

