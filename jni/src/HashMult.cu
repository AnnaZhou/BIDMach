#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <MatKernel.hpp>

#if __CUDA_ARCH__ >= 300
#define MAXXGRID 2147483647
#else
#define MAXXGRID 65535
#endif


// Feature hashing multiply and multiply-transpose.
// This one enumerates, hashes and multiplies all pairs of features.
//
// NOTE: The single-matrix version (hashmult) uses a fast lookup recurrence which is only valid up to 3000 base features per column (approx 4.5 million pairs)


// Hash functions
// Adler32
__forceinline__ __device__ unsigned int adler32(const void *buf, size_t buflength) {
     const unsigned char *buffer = (const unsigned char*)buf;

     unsigned int s1 = 1;
     unsigned int s2 = 0;

     for (size_t n = 0; n < buflength; n++) {
        s1 = (s1 + buffer[n]) % 65521;
        s2 = (s2 + s1) % 65521;
     }     
     return (s2 << 16) | s1;
}

// MurmurHash3

static const unsigned int c1 = 0xcc9e2d51;
static const unsigned int c2 = 0x1b873593;
static const unsigned int r1 = 15;
static const unsigned int r2 = 13;
static const unsigned int m = 5;
static const unsigned int n = 0xe6546b64;

__forceinline__ __device__ unsigned int h1(unsigned int k, unsigned int hash) {

  k *= c1;
  k = (k << r1) | (k >> (32-r1));
  k *= c2;
 
  hash ^= k;
  hash = ((hash << r2) | (hash >> (32-r2)) * m) + n;
  return hash;
}

const unsigned int seed = 3413413;

__forceinline__ __device__ unsigned int mmhashend(unsigned int hash, unsigned int mod)
{
  hash ^= (hash >> 16);
  hash *= 0x85ebca6b;
  hash ^= (hash >> 13);
  hash *= 0xc2b2ae35;
  hash ^= (hash >> 16);
 
  return (hash % mod);
}

__forceinline__ __device__ unsigned int mmhash1(unsigned int v1, unsigned int mod) {
  unsigned int hash = seed;
  hash = h1(v1, hash);
  return mmhashend(hash, mod);
}
  
__forceinline__ __device__ unsigned int mmhash2(unsigned int v1, unsigned int v2, unsigned int mod) {
  unsigned int hash = seed;
  hash = h1(v1, hash);
  hash = h1(v2, hash);
  return mmhashend(hash, mod);
}
  

__forceinline__ __device__ int solve1(int j) {
  float v = sqrtf((float)j);
#pragma unroll
  for (int k = 0; k < 5; k++) {
    v = v - (v*(v+1)-2*j)/(2*v+1);   // Newton iterations to find first index. 
  }
  return (int)(v+2e-5f);   
}

// Given dense A and sparse B, for each column of B, enumerate all pairs of features, hash to a single feature index, and multiply by A into C

__global__ void __hashmult(int nrows, int nfeats, int ncols, int bound1, int bound2, float *A, float *Bdata, int *Bir, int *Bjc, float *C, int transpose) {
  bool doit = false;
  int istart = ((long long)blockIdx.x) * ncols/ gridDim.x;
  int iend = ((long long)(blockIdx.x + 1)) * ncols / gridDim.x;
  for (int i = istart; i < iend ; i++) {                     // i is the column index
    int jstart = Bjc[i];                                     // Range of nz rows in this column
    int jend = Bjc[i+1];
    int nr = jend - jstart;                                  // Number of nz rows
    int todo = nr * (nr + 1) / 2;                            // Number of pairs to process (including k,k pairs)
    for (int j = threadIdx.y; j < todo; j += blockDim.y) {   // j indexes a worker for this column
      int j1 = solve1(j);                                    // Compute the first and second indices
      int j2 = j - j1*(j1+1)/2; 
      float f1 = Bdata[jstart + j1];                         // Get the two features
      float f2 = Bdata[jstart + j2];
      int r1 = Bir[jstart + j1];                             // And their row indices
      int r2 = Bir[jstart + j2];
      int ind = mmhash2(r1, r2, nfeats);                     // Hash the indices
      float prod = f1;
      if (j1 != j2) {
        prod *= f2;
        doit = (r1 < bound1);
      } else {
        long long bigind = ((long long)r1) * r2;
        doit = (bigind < bound2);
      }
      if (doit) {
        if (transpose > 0) {
          float sum = A[threadIdx.x + nrows * i] * f1 * f2;    // Do the product
          atomicAdd(&C[threadIdx.x + nrows * ind], sum);
        } else {
          float sum = A[threadIdx.x + nrows * ind] * f1 * f2;  // Do the product
          atomicAdd(&C[threadIdx.x + nrows * i], sum);
        }
      }
    }
  }
}

int hashmult(int nrows, int nfeats, int ncols, int bound1, int bound2, float *A, float *Bdata, int *Bir, int *Bjc, float *C, int transpose) {
  int nt = max(1, 256/nrows);
  dim3 threadDim(nrows, nt, 1);
  int nblocks = min(MAXXGRID, ncols);
  __hashmult<<<nblocks,threadDim>>>(nrows, nfeats, ncols, bound1, bound2, A, Bdata, Bir, Bjc, C, transpose);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  return err;
}

__global__ void __hashcross(int nrows, int nfeats, int ncols,
			     float *A,
			     float *Bdata, int *Bir, int *Bjc,
			     float *Cdata, int *Cir, int *Cjc,
			     float *D, int transpose) {
  int r1, r2, ind;
  int istart = ((long long)blockIdx.x) * ncols/ gridDim.x;
  int iend = ((long long)(blockIdx.x + 1)) * ncols / gridDim.x;
  for (int i = istart; i < iend ; i++) {                     // i is the column index
    int jstart1 = Bjc[i];                                    // Range of nz rows in this column of B
    int jend1 = Bjc[i+1];
    int jstart2 = Cjc[i];                                    // Range of nz rows in this column of C
    int jend2 = Cjc[i+1];
    int nr1 = jend1 - jstart1;                               // Number of nz rows
    int nr2 = jend2 - jstart2;                               // Number of nz rows
    int todo = (nr1+1) * (nr2+1) - 1;                        // Number of pairs + singletons to process 
    for (int j = threadIdx.y; j < todo; j += blockDim.y) {   // j indexes a worker for this column
      int j1 = j / nr2;
      int j2 = j - j1 * nr2; 
      float prod = 1.0f;
      int hash = seed;
      if (j1 < nr1) {
        prod *= Bdata[jstart1 + j1];                         // Get the two features
        r1 = Bir[jstart1 + j1];                              // And their row indices
        hash = h1(r1, hash);
      }
      if (j2 < nr2) {
        prod *= Cdata[jstart2 + j2];
        r2 = Cir[jstart2 + j2];
        hash = h1(r2, hash);                                 // Hash the indices
      } 
      ind = mmhashend(hash, nfeats);
      if (transpose > 0) {
        float sum = A[threadIdx.x + nrows * i] * prod;       // Do the product
        atomicAdd(&D[threadIdx.x + nrows * ind], sum);
      } else {
        float sum = A[threadIdx.x + nrows * ind] * prod;     
        atomicAdd(&D[threadIdx.x + nrows * i], sum);
      }
    }
  }
}

int hashcross(int nrows, int nfeats, int ncols, float *A, float *Bdata, int *Bir, int *Bjc, float *Cdata, int *Cir, int *Cjc, float *D, int transpose) {
  int nt = max(1, 256/nrows);
  dim3 threadDim(nrows, nt, 1);
  int nblocks = min(MAXXGRID, ncols);
  __hashcross<<<nblocks,threadDim>>>(nrows, nfeats, ncols, A, Bdata, Bir, Bjc, Cdata, Cir, Cjc, D, transpose);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  return err;
}

