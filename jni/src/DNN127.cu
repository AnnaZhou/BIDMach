#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <MatKernel.hpp>


#define NTZ 8
#define NTB (32/NTZ)

#define WLEN 6
#define WLENB 16
#define BYDIM 2

#if __CUDA_ARCH__ >= 300

/*
 *
 * Simple forward convolution kernel for word2vec. Computes the inner products of each column of A with a nearby column of B. 
 * 
 *  SKIP is the max skip-gram length
 *  WINLEN is the length of a block of columns to process
 *
 *  Columns of the output matrix C are <window> (2*SKIP+1) long, contain inner products with corresponding columns of B. 
 *  the row index of C specifies an offset from -SKIP ... SKIP into A, which is the column used for the inner product.
 *  i.e. C(i,j) = <B(:,j), A(:,j-SKIP+i)>
 *
 */

template<int SKIP, int WINLEN, int BDIM>
__global__ void __word2vecFwd(int nrows, int ncols, int *W, float *A, float *B, float *C) {
  const int window = 2*SKIP+1;
  float aa[WINLEN + 2*SKIP];
  float bb[WINLEN];
  float prods[WINLEN][window];
  int word[WINLEN + 2*SKIP];
  __shared__ float CC[WINLEN*BDIM*window];
  int i, j, k, icol;
  int tid = threadIdx.x + blockDim.x * threadIdx.y;
  int dxy = blockDim.x * blockDim.y;
  int istart = (int)((1L * blockIdx.x * ncols) / gridDim.x);
  int iend = (int)((1L * (blockIdx.x+1) * ncols) / gridDim.x);

#pragma unroll
  for (i = 0; i < 2*SKIP; i++) {                           // init context words on edges
    if (i + istart - SKIP > 0) {
      word[i + WINLEN] = W[i + istart - SKIP];
    }
  }

  for (icol = istart; icol < iend; icol += WINLEN) {        // Iterate over columns in blocks of WINLEN
#pragma unroll
    for (j = 0; j < 2*SKIP; j++) {                         // Shift edge words from last time
      word[j] = word[j + WINLEN];
    }
#pragma unroll
    for (i = 0; i < WINLEN; i++) {                          // Get the new words in this window
      if (i + icol + 2*SKIP < ncols) {
        word[j + 2*SKIP] = W[i + icol + 2*SKIP];
      } else {
        word[i + 2*SKIP] = 0;
      }
#pragma unroll
      for (j = 0; j <= 2*SKIP; j++) {                      // clear the products matrix
        prods[i][j] = 0;
      }
    }

    for (i = tid; i < nrows; i += dxy) {                    // Now iterate over the rows of this block
#pragma unroll
      for (j = 0; j < WINLEN + 2*SKIP ; j++) {             // Read A with edges
        aa[j] = A[i + word[j] * nrows];
      }
#pragma unroll
      for (j = 0; j < WINLEN ; j++) {                       // Read B w/o edges, offset by SKIP
        bb[j] = B[i + word[j + SKIP] * nrows];
      }
#pragma unroll
      for (j = 0; j < WINLEN; j++) {                        // Computes the products of these elements
#pragma unroll
        for (k = 0; k <= 2*SKIP; k++) {
          prods[j][k] += aa[j+k] * bb[j];
        }
      }
    }                                                       // Finished the entire block

#pragma unroll
    for (i = 0; i < WINLEN; i++) {                          // Reduce the products within each warp
#pragma unroll
      for (j = 0; j <= 2*SKIP; j++) {
#pragma unroll
        for (k = 1; k < 32; k = k+k) {
          float tmp = __shfl_down(prods[i][j], k);
          prods[i][j] += tmp;
        }
      }
    }

    __syncthreads();
    if (threadIdx.x == 0) {                                 // Save the products to SHMEM (one copy per warp)
#pragma unroll
      for (j = 0; j < WINLEN; j++) {
#pragma unroll
        for (k = 0; k < window; k++) {
          CC[k + window * (j + WINLEN * threadIdx.y)] = prods[j][k];
        }
      }
    }

    __syncthreads();
    for (j = 0; j < WINLEN * window; j += dxy) {            // Reduce the products across warps
      for (k = 1; k < blockDim.y; k++) {
        __syncthreads();
        if (j + tid < WINLEN * window) {
          CC[j + tid] += CC[j + tid + k * WINLEN * window];
        }
      } 
      __syncthreads();
      if (j + tid < WINLEN * window && j + tid + icol * window < iend * window) {
        C[j + tid + icol * window] = CC[j + tid];   // Save the results
      }
    }
    __syncthreads();
  }
}

// Custom convolution kernel for vectors

template<int SKIP>
__global__ void __convRows(int nrows, int ncols, float *A, int lda, float *B, int ldb, float *C) {
  const int window = 2*SKIP+1; 
  const int height = 32 - 2 * SKIP;
  float prods[window];
  int i, j, k;
  int gid = threadIdx.y + blockDim.y * blockIdx.x;
  int tid = threadIdx.x + height * gid;
  float a, b;
  if (tid < nrows) {
#pragma unroll
    for (k = 0; k < window; k++) {
      prods[k] = 0;
    }
    for (i = 0; i < ncols; i++) {
      a = A[tid + i*lda];
      b = B[tid + i*ldb];
#pragma unroll
      for (j = 0; j < height; j++) {
#pragma unroll
        for (k = -SKIP; k <= 0; k++) {
          prods[k+SKIP] += a * __shfl_up(b, -k);
        }
#pragma unroll
        for (k = 1; k <= SKIP; k++) {
          prods[k+SKIP] += a * __shfl_down(b, k);
        }
      }
    }
    if (threadIdx.x >= SKIP && threadIdx.x < 32-SKIP) {
#pragma unroll
      for (k = 0; k < window; k++) {
        C[k + window * (tid - SKIP)] = prods[k];
      }
    }
  }
}

template<int SKIP>
__global__ void __convColsx(int nrows, int ncols, int *W, float *A, float *B, float *C) {
  const int window = 2*SKIP+1; 
  const int width = 32 - 2*SKIP;
  __shared__ float AA[width][33];
  __shared__ float BB[32][33];
  __shared__ float CC[window][33];
  __shared__ int WW[32];
  float prods[window];

  int i, j, k, tid, fid, jcol, word, dxy;
  float a, b;
  dxy = blockDim.x * blockDim.y;
  tid = threadIdx.x + blockDim.x * threadIdx.y;
  fid = tid + dxy * threadIdx.z;
  __syncthreads();                                 
  for (jcol = width * blockIdx.x; jcol < ncols; jcol += width * gridDim.x) {

    __syncthreads();                                 
    if (tid + jcol < ncols) {                             // Load the words for this chunk
      WW[tid] = W[tid + jcol];
    } else {
      WW[tid] = 0;
    }
    __syncthreads();                           

    for (j = threadIdx.z; j < window; j+= blockDim.z) {   // Clear the shared product store
      CC[j][tid] = 0;
    }

    __syncthreads();
#pragma unroll                 
    for (k = 0; k < window; k++) {                        // Clear the register product store
      prods[k] = 0;
    }
    for (i = 0; i < nrows; i += dxy) {                    // process a block of this column
      __syncthreads();
      for (j = threadIdx.z; j < dxy; j += blockDim.z) {   // load data into SHMEM
        word = WW[j];
        if (i + tid < nrows) {
          if (j >= SKIP && j < dxy - SKIP) {
            AA[j-SKIP][tid] = A[i + tid + word * nrows];
          }
          BB[j][tid] = B[i + tid + word * nrows];
        }
      }
      __syncthreads();
#pragma unroll
      for (j = 0; j < NTB; j++) {                         // Get some SHMEM data into registers
        if (tid < width) {
          a = AA[tid][j + NTB*threadIdx.z];
        }
        b = BB[tid][j + NTB*threadIdx.z];
#pragma unroll
        for (k = 0; k < window; k++) {                    // compute shifted products
          prods[k] += a * __shfl_down(b, k);
        }
      }
      __syncthreads();
    }
  
    if (fid < 32) {
#pragma unroll
      for (k = 0; k < window; k++) {                      // move shifted products to SHMEM
        CC[k][tid] = prods[k];
      }
    }
    __syncthreads();
    if (fid >= 32) {
#pragma unroll
      for (k = 0; k < window; k++) {                      // move shifted products to SHMEM
        atomicAdd(&CC[k][tid], prods[k]);
      }
    }
    __syncthreads();                                      // save out to main memory
    if (tid + jcol < ncols) {
      for (i = threadIdx.z; i < window; i += blockDim.z) {
        C[i + (tid + jcol) * window] = CC[i][tid];
      }
    }
    __syncthreads();  
  }
}

#else

template<int SKIP>
__global__ void __convRows(int nrows, int ncols, float *A, int lda, float *B, int ldb, float *C) {}

template<int SKIP, int WINLEN, int BDIM>
__global__ void __word2vecFwd(int nrows, int ncols, int *W, float *A, float *B, float *C) {}


#endif

int convRows(int nrows, int ncols, int shift, float *A, int lda, float *B, int ldb, float *C) {
  dim3 threads(32, 32, 1);
  int nblocks = 1 + (nrows - 1)/threads.y;
  switch(shift) {
  case 5 : __convRows<5><<<nblocks,threads>>>(nrows, ncols, A, lda, B, ldb, C); break;
  case 10 : __convRows<10><<<nblocks,threads>>>(nrows, ncols, A, lda, B, ldb, C); break;
  }
  hipDeviceSynchronize();
  int err = hipGetLastError();
  return err;
}

int word2vecFwd(int nrows, int ncols, int shift, int *W, float *A, float *B, float *C) {
  dim3 threads(32, BYDIM, 1);
  int nblocks = min(4*2048, 2 + (ncols - 1)/WLEN);
  switch(shift) {
  case 5 : __word2vecFwd<5,WLEN,BYDIM><<<nblocks,threads>>>(nrows, ncols, W, A, B, C); break;
    //  case 10 : __convCols<7><<<nblocks,threads>>>(nrows, ncols, W, A, B, C); break;
  }
  hipDeviceSynchronize();
  int err = hipGetLastError();
  return err;
}

