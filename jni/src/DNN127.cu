#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <MatKernel.hpp>

#define BYDIM 2

#if __CUDA_ARCH__ >= 300

template<int NSKIP, int NNEG, int NELTS, int NYDIM>
  __global__ void __word2vec(int nrows, int ncols, int *WA, int *WB, float *A, float *B, float lrate) {
  const int NWINDOW = 1 + 2 * NSKIP;
  float aa[NELTS][NWINDOW];
  float daa[NELTS][NWINDOW];
  float bb[NELTS];
  float dbb[NELTS];
  __shared__ float prods[NYDIM][NNEG*NWINDOW];
  __shared__ int wb[NNEG*NWINDOW];

  int tid = threadIdx.x + blockDim.x * threadIdx.y;
  int dxy = blockDim.x * blockDim.y;
  int i, j, k, icol, jneg, thiscol, wa;
  float f, g, label;
  int istart = (int)((1L * blockIdx.x * ncols) / gridDim.x);
  int iend = (int)((1L * (blockIdx.x+1) * ncols) / gridDim.x);
  bool good = false;

#pragma unroll
  for (icol = 0; icol < NSKIP; icol++) {                  // Fill up the data WINDOW
    thiscol = istart + icol + 1;
    good = (thiscol > 0 && thiscol < ncols);
    if (good) {
      wa = WA[thiscol];
    } else {
      wa = 0;
    }
#pragma unroll
    for (i = 0; i < NELTS; i++) {                           // get the column data in NELTS sections
      if (good && tid + i*dxy < nrows) {
        aa[i][icol+NSKIP+1] = A[tid + i*dxy + wa * nrows];          // Load the new data
      } else {
        aa[i][icol+NSKIP+1] = 0;
      }
    }
  }

  for (icol = istart; icol < iend; icol++) {                // Iterate over columns

    // Load the last column in the window into register memory
    thiscol = icol + NSKIP;
    good = (thiscol < ncols);
    if (good) {
      wa = WA[thiscol];                                     // get the word index
    } else {
      wa = 0;
    }
#pragma unroll
    for (i = 0; i < NELTS; i++) {                           // get the column data in NELTS sections
#pragma unroll
      for (j = 0; j < NWINDOW-1; j++) {                     // Need to shift the saved data (register arrays not indexable)
        aa[i][j] = aa[i][j+1];
        daa[i][j] = daa[i][j+1];
      }
    }
#pragma unroll
    for (i = 0; i < NELTS; i++) {                           // get the column data in NELTS sections
      if (good && tid + i*dxy < nrows) {
        aa[i][NWINDOW-1] = A[tid + i*dxy + wa * nrows];     // Load the new data
      } else {
        aa[i][NWINDOW-1] = 0;   
      }
      daa[i][NWINDOW-1] = 0;                                // Clear the derivative
    }

    // Get negative column indices
    __syncthreads();
    if (tid < NNEG*NWINDOW) {                                  
      wb[tid] = WB[tid + NNEG * NWINDOW * icol];
    }
    __syncthreads();
      // Compute all the inner products with the current negative
#pragma unroll
    for (j = 0; j < NWINDOW; j++) {     
    // Iterate over the negatives
#pragma unroll
      for (jneg = 0; jneg < NNEG; jneg++) {                   // Iterate over the negatives
#pragma unroll
        for (i = 0; i < NELTS; i++) {                         // load the current negative column in NELTS sections
          if (tid + i*dxy < nrows) {
            bb[i] = B[tid + i*dxy + wb[jneg + NNEG*j] * nrows]; 
          } else {
            bb[i] = 0;
          }
          dbb[i] = 0;
        }

        f = 0;
#pragma unroll
        for (i = 0; i < NELTS; i++) {                         // load the current negative column in NELTS sections
          f += aa[i][j] * bb[i];                            // partial product
        }
        // This section reduces f over the column
#pragma unroll
        for (k = 1; k < 32; k = k+k) {                      // Reduce f in a warp
          float tmp = __shfl_down(f, k);
          f += tmp;
        }
        __syncthreads();
        if (threadIdx.x == 0) {                             // Save f to SHMEM
          prods[threadIdx.y][0] = f;
        }
        __syncthreads();
        if (tid == 0) {
          for (i = 1; i < NYDIM; i++) {                         // Reduce in SHMEM 
            prods[0][0] += prods[i][0];
          }
        }
        __syncthreads();
        f = prods[0][0];
        // Compute g from f
        label = (jneg == 0);
        if (f > 12.0f) {
          g = 1.0f;
        } else {
          float expf = exp(f);
          g = expf / (1.0f + expf);
        } 
        g = (label - g) * lrate;

#pragma unroll
        for (i = 0; i < NELTS; i++) {     
          daa[i][j] += g * bb[i];
          dbb[i] += g * aa[i][j];
        }
#pragma unroll
        for (i = 0; i < NELTS; i++) {                         // Save the update to the negative column
          if (tid + i*dxy < nrows) {
            atomicAdd(&B[tid + i*dxy + wb[jneg] * nrows], dbb[i]);
          }
        }
      }
    } 
    __syncthreads();
    thiscol = icol - NSKIP;
    if (thiscol >= 0 && thiscol < ncols) {
      wa = WA[thiscol];                                     // get the word index
#pragma unroll
      for (i = 0; i < NELTS; i++) {                 
        if (tid + i*dxy < nrows) {
          atomicAdd(&A[tid + i*dxy + wa * nrows], daa[i][0]);
        }
      }
    }
  }
}

/*
 *
 * Simple forward convolution kernel for word2vec. Computes inner products of columns from A with columns from B. 
 * The column indices are specified by two "word" matrices. The inner products are computed as an outer product
 * of the word matrices.
 * 
 *  SKIP is the max skip-gram length
 *  WINLEN is the length of a block of columns to process
 *
 *  Columns of the output matrix C are <window> = 2*SKIP+1 long, and contain inner products with corresponding columns of B. 
 *  the row index of C specifies an offset from -SKIP ... SKIP into A, which is the column used for the inner product.
 *  i.e. C(i,j) = <B(:,j), A(:,j-SKIP+i)>
 *
 */

template<int NWA, int NWB, int BDIM>
  __global__ void __word2vecFwd(int nrows, int ncols, int *WA, int *WB, float *A, float *B, float *C) {
  const int nwab = NWA*NWB;
  __shared__ float CC[NWA*NWB*BDIM];
  float aa[NWA];
  float bb[NWB];
  float prods[NWA][NWB];
  int wa[NWA];
  int wb[NWB];
  int tid = threadIdx.x + blockDim.x * threadIdx.y;
  int dxy = blockDim.x * blockDim.y;
  int i, j, k, icol;
  int istart = (int)((1L * blockIdx.x * ncols) / gridDim.x);
  int iend = (int)((1L * (blockIdx.x+1) * ncols) / gridDim.x);

  for (icol = istart; icol < iend; icol++) {            // Iterate over columns
#pragma unroll
    for (i = 0; i < NWA; i++) {
      for (j = 0; j < NWB; j++) {                       // clear the products matrix
        prods[i][j] = 0;
      }
      wa[i] = WA[i + icol * NWA];                       // Fill the A word matrix
    }
    for (i = 0; i < NWB; i++) {
      wb[i] = WB[i + icol * NWB];                       // Fill the B word matrix
    }

    for (i = tid; i < nrows; i += dxy) {                // Now iterate over the rows of this block
#pragma unroll
      for (j = 0; j < NWA; j++) {                       // Read A
        aa[j] = A[i + wa[j] * nrows];
      }
#pragma unroll
      for (j = 0; j < NWB ; j++) {                      // Read B
        bb[j] = B[i + wb[j] * nrows];
      }
#pragma unroll
      for (j = 0; j < NWA; j++) {                        // Computes the products of these elements
#pragma unroll
        for (k = 0; k < NWB; k++) {
          prods[j][k] += aa[j] * bb[k];
        }
      }
    }                                                    // Finished the entire block

#pragma unroll
    for (i = 0; i < NWA; i++) {                          // Reduce the products within each warp
#pragma unroll
      for (j = 0; j < NWB; j++) {
#pragma unroll
        for (k = 1; k < 32; k = k+k) {
          float tmp = __shfl_down(prods[i][j], k);
          prods[i][j] += tmp;
        }
      }
    }

    __syncthreads();
    if (threadIdx.x == 0) {                               // Save the products to SHMEM (one copy per warp)
#pragma unroll
      for (i = 0; i < NWA; i++) {
#pragma unroll
        for (j = 0; j < NWB; j++) {
          CC[j + NWB * (i + NWA * threadIdx.y)] = prods[i][j];
        }
      }
    }

    __syncthreads();
    for (i = 1; i < blockDim.y; i++) {
      __syncthreads();
      for (j = tid; j < nwab; j += dxy) {                   // Reduce the products across warps
        CC[j] += CC[j + i * nwab];
      } 
    } 
    __syncthreads();
    for (i = tid; i < nwab; i += dxy) {                     // Save to main memory
      C[i + icol * nwab] = CC[i];  
        //atomicAdd(&C[i + icol * nwab], CC[i]); 
    }
    __syncthreads();
  }
}

/*
 *
 * Simple forward convolution kernel for word2vec. Computes the inner products of each column of A with a nearby column of B. 
 * 
 *  SKIP is the max skip-gram length
 *  WINLEN is the length of a block of columns to process
 *
 *  Columns of the output matrix C are <window> = 2*SKIP+1 long, and contain inner products with corresponding columns of B. 
 *  the row index of C specifies an offset from -SKIP ... SKIP into A, which is the column used for the inner product.
 *  i.e. C(i,j) = <B(:,j), A(:,j-SKIP+i)>
 *
 */

template<int SKIP, int WINLEN, int BDIM>
__global__ void __word2vecFwdx(int nrows, int ncols, int *W, float *A, float *B, float *C) {
  const int window = 2*SKIP+1;
  float aa[WINLEN + 2*SKIP];
  float bb[WINLEN];
  float prods[WINLEN][window];
  int word[WINLEN + 2*SKIP];
  __shared__ float CC[WINLEN*BDIM*window];
  int i, j, k, icol;
  int tid = threadIdx.x + blockDim.x * threadIdx.y;
  int dxy = blockDim.x * blockDim.y;
  int istart = (int)((1L * blockIdx.x * ncols) / gridDim.x);
  int iend = (int)((1L * (blockIdx.x+1) * ncols) / gridDim.x);

#pragma unroll
  for (i = 0; i < 2*SKIP; i++) {                            // init context words on edges
    if (i + istart - SKIP > 0) {
      word[i + WINLEN] = W[i + istart - SKIP];
    }
  }

  for (icol = istart; icol < iend; icol += WINLEN) {        // Iterate over columns in blocks of WINLEN
#pragma unroll
    for (j = 0; j < 2*SKIP; j++) {                          // Shift edge words from last time
      word[j] = word[j + WINLEN];
    }
#pragma unroll
    for (i = 0; i < WINLEN; i++) {                          // Get the new words in this window
      if (i + icol + 2*SKIP < ncols) {
        word[j + 2*SKIP] = W[i + icol + 2*SKIP];
      } else {
        word[i + 2*SKIP] = 0;
      }
#pragma unroll
      for (j = 0; j <= 2*SKIP; j++) {                       // clear the products matrix
        prods[i][j] = 0;
      }
    }

    for (i = tid; i < nrows; i += dxy) {                    // Now iterate over the rows of this block
#pragma unroll
      for (j = 0; j < WINLEN + 2*SKIP ; j++) {              // Read A with edges
        aa[j] = A[i + word[j] * nrows];
      }
#pragma unroll
      for (j = 0; j < WINLEN ; j++) {                       // Read B w/o edges, offset by SKIP
        bb[j] = B[i + word[j + SKIP] * nrows];
      }
#pragma unroll
      for (j = 0; j < WINLEN; j++) {                        // Computes the products of these elements
#pragma unroll
        for (k = 0; k <= 2*SKIP; k++) {
          prods[j][k] += aa[j+k] * bb[j];
        }
      }
    }                                                       // Finished the entire block

#pragma unroll
    for (i = 0; i < WINLEN; i++) {                          // Reduce the products within each warp
#pragma unroll
      for (j = 0; j <= 2*SKIP; j++) {
#pragma unroll
        for (k = 1; k < 32; k = k+k) {
          float tmp = __shfl_down(prods[i][j], k);
          prods[i][j] += tmp;
        }
      }
    }

    __syncthreads();
    if (threadIdx.x == 0) {                                 // Save the products to SHMEM (one copy per warp)
#pragma unroll
      for (j = 0; j < WINLEN; j++) {
#pragma unroll
        for (k = 0; k < window; k++) {
          CC[k + window * (j + WINLEN * threadIdx.y)] = prods[j][k];
        }
      }
    }

    __syncthreads();
    for (j = 0; j < WINLEN * window; j += dxy) {            // Reduce the products across warps
      for (k = 1; k < blockDim.y; k++) {
        __syncthreads();
        if (j + tid < WINLEN * window) {
          CC[j + tid] += CC[j + tid + k * WINLEN * window];
        }
      } 
      __syncthreads();
      if (j + tid < WINLEN * window && j + tid + icol * window < iend * window) {
        C[j + tid + icol * window] = CC[j + tid];   // Save the results
      }
    }
    __syncthreads();
  }
}


// Custom convolution kernel for vectors

template<int SKIP>
__global__ void __convRows(int nrows, int ncols, float *A, int lda, float *B, int ldb, float *C) {
  const int window = 2*SKIP+1; 
  const int height = 32 - 2 * SKIP;
  float prods[window];
  int i, j, k;
  int gid = threadIdx.y + blockDim.y * blockIdx.x;
  int tid = threadIdx.x + height * gid;
  float a, b;
  if (tid < nrows) {
#pragma unroll
    for (k = 0; k < window; k++) {
      prods[k] = 0;
    }
    for (i = 0; i < ncols; i++) {
      a = A[tid + i*lda];
      b = B[tid + i*ldb];
#pragma unroll
      for (j = 0; j < height; j++) {
#pragma unroll
        for (k = -SKIP; k <= 0; k++) {
          prods[k+SKIP] += a * __shfl_up(b, -k);
        }
#pragma unroll
        for (k = 1; k <= SKIP; k++) {
          prods[k+SKIP] += a * __shfl_down(b, k);
        }
      }
    }
    if (threadIdx.x >= SKIP && threadIdx.x < 32-SKIP) {
#pragma unroll
      for (k = 0; k < window; k++) {
        C[k + window * (tid - SKIP)] = prods[k];
      }
    }
  }
}

template<int SKIP>
__global__ void __convColsx(int nrows, int ncols, int *W, float *A, float *B, float *C) {
  const int window = 2*SKIP+1; 
  const int width = 32 - 2*SKIP;
  __shared__ float AA[width][33];
  __shared__ float BB[32][33];
  __shared__ float CC[window][33];
  __shared__ int WW[32];
  float prods[window];

  int i, j, k, tid, fid, jcol, word, dxy;
  float a, b;
  dxy = blockDim.x * blockDim.y;
  tid = threadIdx.x + blockDim.x * threadIdx.y;
  fid = tid + dxy * threadIdx.z;
  __syncthreads();                                 
  for (jcol = width * blockIdx.x; jcol < ncols; jcol += width * gridDim.x) {

    __syncthreads();                                 
    if (tid + jcol < ncols) {                             // Load the words for this chunk
      WW[tid] = W[tid + jcol];
    } else {
      WW[tid] = 0;
    }
    __syncthreads();                           

    for (j = threadIdx.z; j < window; j+= blockDim.z) {   // Clear the shared product store
      CC[j][tid] = 0;
    }

    __syncthreads();
#pragma unroll                 
    for (k = 0; k < window; k++) {                        // Clear the register product store
      prods[k] = 0;
    }
    for (i = 0; i < nrows; i += dxy) {                    // process a block of this column
      __syncthreads();
      for (j = threadIdx.z; j < dxy; j += blockDim.z) {   // load data into SHMEM
        word = WW[j];
        if (i + tid < nrows) {
          if (j >= SKIP && j < dxy - SKIP) {
            AA[j-SKIP][tid] = A[i + tid + word * nrows];
          }
          BB[j][tid] = B[i + tid + word * nrows];
        }
      }
      __syncthreads();
#pragma unroll
      for (j = 0; j < NTB; j++) {                         // Get some SHMEM data into registers
        if (tid < width) {
          a = AA[tid][j + NTB*threadIdx.z];
        }
        b = BB[tid][j + NTB*threadIdx.z];
#pragma unroll
        for (k = 0; k < window; k++) {                    // compute shifted products
          prods[k] += a * __shfl_down(b, k);
        }
      }
      __syncthreads();
    }
  
    if (fid < 32) {
#pragma unroll
      for (k = 0; k < window; k++) {                      // move shifted products to SHMEM
        CC[k][tid] = prods[k];
      }
    }
    __syncthreads();
    if (fid >= 32) {
#pragma unroll
      for (k = 0; k < window; k++) {                      // move shifted products to SHMEM
        atomicAdd(&CC[k][tid], prods[k]);
      }
    }
    __syncthreads();                                      // save out to main memory
    if (tid + jcol < ncols) {
      for (i = threadIdx.z; i < window; i += blockDim.z) {
        C[i + (tid + jcol) * window] = CC[i][tid];
      }
    }
    __syncthreads();  
  }
}

/*
 *
 * Simple backward convolution kernel for word2vec. 
 * Computes the gradient for A given B or vice-versa, and does an SGD update.
 * 
 *  SKIP is the max skip-gram length
 *  WINLEN is the length of a block of columns to process 
 *
 */


template<int NWA, int NWB, int MAXDIM>
  __global__ void __word2vecBwd(int nrows, int ncols, int *WA, int *WB, float *A, float *B, float *C, float lrate) {
  const int nwab = NWA * NWB;
  float dd[MAXDIM];
  int wa[NWA];
  int wb[NWB];
  __shared__ float cc[NWA*NWB];
  int tid = threadIdx.x;
  int fid = threadIdx.x + blockDim.x * threadIdx.y; 
  int dxy = blockDim.x * blockDim.y;
  int icol, i, j, k;
  float sum;
  int istart = (int)((1L * blockIdx.x * ncols) / gridDim.x);
  int iend = (int)((1L * (blockIdx.x+1) * ncols) / gridDim.x);

  for (icol = istart; icol < iend; icol++) {            // iterate in columns
#pragma unroll
    for (j = 0; j < NWA; j++) {
      wa[j] = WA[j + icol * NWA];                       // Load the A word matrix
    }
    __syncthreads();
#pragma unroll 
    for (j = 0; j < NWB; j++) {
      wb[j] = WB[j + icol * NWB];                       // Load the B word matrix
    }
    for (i = fid; i < nwab; i += dxy) {
      cc[i] = C[i + icol * nwab];
    }
    __syncthreads();
    for (i = tid; i < nrows; i += dxy) {
#pragma unroll
      for (j = 0; j < NWB; j++) {                       // Load the data
        dd[j] = B[i + wb[j] * nrows];
      }

      for (j = 0; j < NWA; j++) {                         // Now do the product
        sum = 0;
#pragma unroll
        for (k = 0; k < NWB; k++) {                       
          float xx =  cc[j + k * NWA];
          sum += xx * dd[k];
        }
        atomicAdd(&A[i + wa[j] * nrows], sum * lrate);
      }

#pragma unroll
      for (j = 0; j < NWA; j++) {                       // Load the data
        dd[j] = A[i + wa[j] * nrows];
      }

      for (j = 0; j < NWB; j++) {                         // Now do the product
        sum = 0;
#pragma unroll
        for (k = 0; k < NWA; k++) {                       
          float xx =  cc[k + j * NWA];
          sum += xx * dd[k];
        }
        atomicAdd(&B[i + wb[j] * nrows], sum * lrate);
      }
    }
  }
}

#else

template<int NSKIP, int NNEG, int NELTS, int NYDIM>
  __global__ void __word2vec(int nrows, int ncols, int *WA, int *WB, float *A, float *B, float lrate) {}

template<int SKIP>
__global__ void __convRows(int nrows, int ncols, float *A, int lda, float *B, int ldb, float *C) {}

template<int NWA, int NWB, int BDIM>
__global__ void __word2vecFwd(int nrows, int ncols, int *WA, int *WB, float *A, float *B, float *C) {}

template<int NWA, int NWB, int MAXDIM>
  __global__ void __word2vecBwd(int nrows, int ncols, int *WA, int *WB, float *A, float *B, float *C, float lrate) {}


#endif

int word2vec(int nrows, int ncols, int *WA, int *WB, float *A, float *B, float lrate) {
  const int NSKIP = 5;
  const int NNEG = 5;
  const int NELTS = 5;
  const int NYDIM = 2; 
  dim3 threads(32, NYDIM, 1);
  int nblocks = min(1024, 2 + (ncols - 1));
  __word2vec<NSKIP,NNEG,NELTS,NYDIM><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, lrate);
  hipDeviceSynchronize(); 
  int err = hipGetLastError();
  return err;
}

int convRows(int nrows, int ncols, int shift, float *A, int lda, float *B, int ldb, float *C) {
  dim3 threads(32, 32, 1);
  int nblocks = 1 + (nrows - 1)/threads.y;
  switch(shift) {
  case 5 : __convRows<5><<<nblocks,threads>>>(nrows, ncols, A, lda, B, ldb, C); break;
  case 10 : __convRows<10><<<nblocks,threads>>>(nrows, ncols, A, lda, B, ldb, C); break;
  }
  hipDeviceSynchronize();
  int err = hipGetLastError();
  return err;
}

int word2vecFwd(int nrows, int ncols, int nwa, int nwb, int *WA, int *WB, float *A, float *B, float *C) {
  dim3 threads(32, BYDIM, 1);
  int nblocks = min(4096, 2 + (ncols - 1));
  int which = nwa*10000 + nwb;
  switch (which) {
  case 10005: __word2vecFwd<1,5,BYDIM><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, C); break;
  case 50005: __word2vecFwd<5,5,BYDIM><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, C); break;
  case 110005: __word2vecFwd<11,5,BYDIM><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, C); break;
  case 80006: __word2vecFwd<8,6,BYDIM><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, C); break;
  default : printf("word2vecFwd unsupport size combination %d %d\n", nwa, nwb); return 1;
  }
  hipDeviceSynchronize();
  int err = hipGetLastError();
  return err;
  }

int word2vecBwd(int nrows, int ncols, int nwa, int nwb, int *WA, int *WB, float *A, float *B, float *C, float lrate) {
  dim3 threads(32*BYDIM, 1, 1);
  int nblocks = min(2048, 2 + (ncols - 1));
  int which = nwa*10000 + nwb;
  switch (which) {
  case 10005: __word2vecBwd<1,5,5><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, C, lrate); break;
  case 50005: __word2vecBwd<5,5,5><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, C, lrate); break;
  case 110005: __word2vecBwd<11,5,11><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, C, lrate); break;
  case 80006: __word2vecBwd<8,6,8><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, C, lrate); break;
  default : printf("word2vecBwd unsupport size combination %d %d\n", nwa, nwb); return 1;
  }
  hipDeviceSynchronize();
  int err = hipGetLastError();
  return err;
}
