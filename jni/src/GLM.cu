#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <MatKernel.hpp>

typedef float (*fntype)(float);
typedef float (*optype)(float,float);

__device__ float link_linear(float a) {return a;}
__device__ float link_logistic(float a) {return log(a/(1.0f - a));}

__device__ float mean_linear(float a) {return a;}
__device__ float mean_logistic(float a) {
  float tmp;
  if (a > 0) {
    tmp = exp(-a);
    return 1.0f/(1.0f + tmp);
  } else {
    tmp = exp(a);
    return tmp/(1.0f + tmp);
  }
}

__device__ float deriv_linear(float a, float b) {return b-a;}
__device__ float deriv_logistic(float a, float b) {return b-a;}
__device__ float deriv_maxp(float p, float t) {return (2.0f*t - 1.0f)*p*(1.0f-p);}
__device__ float deriv_svm(float p, float t) {
  float tt = 2 * t - 1;
  return (p * tt < 1.0f) ? tt : 0.0f;
}


#define eps 1.0e-10f
__device__ float ll_linear(float a, float t) {return (t-a)*(a-t);}
__device__ float ll_logistic(float a, float b) {return log(a * b + (1.0f - a) * (1.0f - b) + eps);}
__device__ float ll_maxp(float a, float t) {return a * t + (1.0f - a) * (1.0f - t) - 1.0f;}
__device__ float ll_svm(float p, float t) {
  float tt = 2 * t - 1;
  return min(0.0f, tt * p - 1);
}

__device__ const fntype linkfns[] = {
  link_linear,
  link_logistic,
  link_logistic,
  link_linear};

__device__ const fntype meanfns[] = {
  mean_linear,
  mean_logistic,
  mean_logistic,
  mean_linear};

__device__ const optype derivfns[] = {
  deriv_linear,
  deriv_logistic,
  deriv_maxp,
  deriv_svm};

__device__ const optype llfns[] = {
  ll_linear,
  ll_logistic,
  ll_maxp,
  ll_svm};


typedef double (*dfntype)(double);
typedef double (*doptype)(double,double);

__device__ double dlink_linear(double a) {return a;}
__device__ double dlink_logistic(double a) {return log(a/(1.0 - a));}

__device__ double dmean_linear(double a) {return a;}
__device__ double dmean_logistic(double a) {
  double tmp;
  if (a > 0) {
    tmp = exp(-a);
    return 1.0/(1.0 + tmp);
  } else {
    tmp = exp(a);
    return tmp/(1.0 + tmp);
  }
}

__device__ double dderiv_linear(double a, double b) {return b-a;}
__device__ double dderiv_logistic(double a, double b) {return b-a;}
__device__ double dderiv_maxp(double p, double t) {return (2.0*t - 1.0f)*p*(1.0-p);}
__device__ double dderiv_svm(double p, double t) {
  double tt = 2 * t - 1;
  return (p * tt < 1.0) ? tt : 0.0;
}


#define eps 1.0e-10f
__device__ double dll_linear(double a, double t) {return (t-a)*(a-t);}
__device__ double dll_logistic(double a, double b) {return log(a * b + (1.0 - a) * (1.0 - b) + eps);}
__device__ double dll_maxp(double a, double t) {return a * t + (1.0 - a) * (1.0 - t) - 1.0;}
__device__ double dll_svm(double p, double t) {
  double tt = 2 * t - 1;
  return min(0.0, tt * p - 1);
}

__device__ const dfntype dlinkfns[] = {
  dlink_linear,
  dlink_logistic,
  dlink_logistic,
  dlink_linear};

__device__ const dfntype dmeanfns[] = {
  dmean_linear,
  dmean_logistic,
  dmean_logistic,
  dmean_linear};

__device__ const doptype dderivfns[] = {
  dderiv_linear,
  dderiv_logistic,
  dderiv_maxp,
  dderiv_svm};

__device__ const doptype dllfns[] = {
  dll_linear,
  dll_logistic,
  dll_maxp,
  dll_svm};


void setsizes(int N, dim3 *gridp, int *nthreadsp) {
  int nblocks = 1;
  int nthreads = 1;
  while (nblocks * nthreads < N) {
    if (nblocks < 16) {
      nblocks = 2*nblocks;
    } else if (nthreads < 1024) {
      nthreads = 2*nthreads;
    } else {
      nblocks = 2*nblocks;
    }
  }
  gridp->y = 1 + (nblocks-1)/65536;
  gridp->x = 1 + (nblocks-1)/gridp->y;
  gridp->z = 1;
  *nthreadsp = nthreads;
}

__global__ void __apply_preds(float *A, int *L, float *C, int nrows, int ncols) {
  int ip = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
  for (int i = ip; i < nrows*ncols; i += blockDim.x * gridDim.x * gridDim.y) {
    fntype fn = meanfns[L[i % nrows]];
    C[i] = fn(A[i]);
  }
}

int apply_preds(float *A, int *L, float *C, int nrows, int ncols) {
  int nthreads;
  dim3 griddims;
  setsizes(nrows*ncols, &griddims, &nthreads);
  __apply_preds<<<griddims,nthreads>>>(A, L, C, nrows, ncols);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  return err;
}

__global__ void __apply_links(float *A, int *L, float *C, int nrows, int ncols) {
  int ip = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
  for (int i = ip; i < nrows*ncols; i += blockDim.x * gridDim.x * gridDim.y) {
    fntype fn = linkfns[L[i % nrows]];
    C[i] = fn(A[i]);
  }
}

int apply_links(float *A, int *L, float *C, int nrows, int ncols) {
  int nthreads;
  dim3 griddims;
  setsizes(nrows*ncols, &griddims, &nthreads);
  __apply_links<<<griddims,nthreads>>>(A, L, C, nrows, ncols);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  return err;
}

__global__ void __apply_lls(float *A, float *B, int *L, float *C, int nrows, int ncols) {
  int ip = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
  for (int i = ip; i < nrows*ncols; i += blockDim.x * gridDim.x * gridDim.y) {
    optype op = llfns[L[i % nrows]];
    C[i] = op(A[i],B[i]);
  }
}


int apply_lls(float *A, float *B, int *L, float *C, int nrows, int ncols) {
  int nthreads;
  dim3 griddims;
  setsizes(nrows*ncols, &griddims, &nthreads);
  __apply_lls<<<griddims,nthreads>>>(A, B, L, C, nrows, ncols);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  return err;
}

__global__ void __apply_derivs(float *A, float *B, int *L, float *C, int nrows, int ncols) {
  int ip = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
  for (int i = ip; i < nrows*ncols; i += blockDim.x * gridDim.x * gridDim.y) {
    optype op = derivfns[L[i % nrows]];
    C[i] = op(A[i],B[i]);
  }
}

int apply_derivs(float *A, float *B, int *L, float *C, int nrows, int ncols) {
  int nthreads;
  dim3 griddims;
  setsizes(nrows*ncols, &griddims, &nthreads);
  __apply_derivs<<<griddims,nthreads>>>(A, B, L, C, nrows, ncols);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  return err;
}

__global__ void __apply_dpreds(double *A, int *L, double *C, int nrows, int ncols) {
  int ip = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
  for (int i = ip; i < nrows*ncols; i += blockDim.x * gridDim.x * gridDim.y) {
    dfntype fn = dmeanfns[L[i % nrows]];
    C[i] = fn(A[i]);
  }
}

int apply_dpreds(double *A, int *L, double *C, int nrows, int ncols) {
  int nthreads;
  dim3 griddims;
  setsizes(nrows*ncols, &griddims, &nthreads);
  __apply_dpreds<<<griddims,nthreads>>>(A, L, C, nrows, ncols);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  return err;
}

__global__ void __apply_dlinks(double *A, int *L, double *C, int nrows, int ncols) {
  int ip = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
  for (int i = ip; i < nrows*ncols; i += blockDim.x * gridDim.x * gridDim.y) {
    dfntype fn = dlinkfns[L[i % nrows]];
    C[i] = fn(A[i]);
  }
}

int apply_dlinks(double *A, int *L, double *C, int nrows, int ncols) {
  int nthreads;
  dim3 griddims;
  setsizes(nrows*ncols, &griddims, &nthreads);
  __apply_dlinks<<<griddims,nthreads>>>(A, L, C, nrows, ncols);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  return err;
}

__global__ void __apply_dlls(double *A, double *B, int *L, double *C, int nrows, int ncols) {
  int ip = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
  for (int i = ip; i < nrows*ncols; i += blockDim.x * gridDim.x * gridDim.y) {
    doptype op = dllfns[L[i % nrows]];
    C[i] = op(A[i],B[i]);
  }
}


int apply_dlls(double *A, double *B, int *L, double *C, int nrows, int ncols) {
  int nthreads;
  dim3 griddims;
  setsizes(nrows*ncols, &griddims, &nthreads);
  __apply_dlls<<<griddims,nthreads>>>(A, B, L, C, nrows, ncols);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  return err;
}

__global__ void __apply_dderivs(double *A, double *B, int *L, double *C, int nrows, int ncols) {
  int ip = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
  for (int i = ip; i < nrows*ncols; i += blockDim.x * gridDim.x * gridDim.y) {
    doptype op = dderivfns[L[i % nrows]];
    C[i] = op(A[i],B[i]);
  }
}

int apply_dderivs(double *A, double *B, int *L, double *C, int nrows, int ncols) {
  int nthreads;
  dim3 griddims;
  setsizes(nrows*ncols, &griddims, &nthreads);
  __apply_dderivs<<<griddims,nthreads>>>(A, B, L, C, nrows, ncols);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  return err;
}
